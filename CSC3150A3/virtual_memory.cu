﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <iostream>
#include <list>

__device__ void init_invert_page_table(VirtualMemory *vm) {

  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    vm->invert_page_table[i] = 0x80000000; // invalid := MSB is 1
    vm->invert_page_table[i + vm->PAGE_ENTRIES] = i;
  }
}

__device__ void vm_init(VirtualMemory *vm, uchar *buffer, uchar *storage,
                        u32 *invert_page_table, int *pagefault_num_ptr,
                        int PAGESIZE, int INVERT_PAGE_TABLE_SIZE,
                        int PHYSICAL_MEM_SIZE, int STORAGE_SIZE,
                        int PAGE_ENTRIES) {
  // init variables
  vm->buffer = buffer;
  vm->storage = storage;
  vm->invert_page_table = invert_page_table;
  vm->pagefault_num_ptr = pagefault_num_ptr;

  // init constants
  vm->PAGESIZE = PAGESIZE;
  vm->INVERT_PAGE_TABLE_SIZE = INVERT_PAGE_TABLE_SIZE;
  vm->PHYSICAL_MEM_SIZE = PHYSICAL_MEM_SIZE;
  vm->STORAGE_SIZE = STORAGE_SIZE;
  vm->PAGE_ENTRIES = PAGE_ENTRIES;

  // before first vm_write or vm_read
  init_invert_page_table(vm);
}

__device__ uchar vm_read(VirtualMemory *vm, u32 addr) {
  /* Complate vm_read function to read single element from data buffer */

	printf("vm_read\n");
	int page_offset = addr % 32;
	int page_num = addr / 32;
	u32 current = vm->invert_page_table[page_num];
	printf("frame = %ld\n", current);
	int frame_num = vm->invert_page_table[page_num + vm->PAGE_ENTRIES];
	u32 phy_addr = frame_num * vm->PAGESIZE + page_offset;
	//if (current = 0x8000000) {
	//	*vm->pagefault_num_ptr += 1;
	//}
	//else {
	//	 
	//}

	
}

__device__ void vm_write(VirtualMemory *vm, u32 addr, uchar value) {
  /* Complete vm_write function to write value into data buffer */
	int page_offset = addr % 32;
	int page_num = addr / 32;
	u32 current = vm->invert_page_table[page_num];  // get current valid/invalid code
	int frame_num = vm->invert_page_table[page_num + vm->PAGE_ENTRIES];
	u32 phy_addr = frame_num * vm->PAGESIZE + page_offset;
	printf("curr frame = %ld\n", frame_num);
	if (current == 0x80000000) {
		*vm->pagefault_num_ptr += 1; // add one more page fault
		/*printf("fault num = %d\n", vm->pagefault_num_ptr);*/	
		vm->invert_page_table[page_num] = page_num;
	} 
	vm->buffer[phy_addr] = value;
	printf("write %c\n", *(vm->buffer + phy_addr));
	vm->invert_page_table[page_num + vm->PAGE_ENTRIES] = frame_num;

	//printf("offset = %ld\n", page_offset);
	//printf("page_num = %ld\n", page_num);
	//printf("current phy_addr = %08" PRIx32 "\n", current);


  


}

__device__ void vm_snapshot(VirtualMemory *vm, uchar *results, int offset,
                            int input_size) {
  /* Complete snapshot function togther with vm_read to load elements from data
   * to result buffer */
	
}

